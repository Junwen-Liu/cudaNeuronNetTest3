#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "sigmoid_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>

__device__ float sigmoid(float x) {
	return 1.0f / (1 + exp(-x));
}

//Z is layer's input, A is layer's output
__global__ void sigmoidActivationForward(float* Z, float* A,
										 int Z_x_dim, int Z_y_dim) {
	//calculate index for current thread
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		A[index] = sigmoid(Z[index]);
	}
}

//dA is the error introduced by this layer
__global__ void sigmoidActivationBackprop(float* Z, float* dA, float* dZ,
										  int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		dZ[index] = dA[index] * sigmoid(Z[index]) * (1 - sigmoid(Z[index]));
	}
}

SigmoidActivation::SigmoidActivation(std::string name) {
	//to access members of a structure through pointer, use arrow operator, shorthand for (*this).name
	this->name = name;
}

SigmoidActivation::~SigmoidActivation()
{ }

Matrix& SigmoidActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	sigmoidActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(), A.data_device.get(),
														   	Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform sigmoid forward propagation.");

	return A;
}

Matrix& SigmoidActivation::backprop(Matrix& dA, float learning_rate) {
	dZ.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	sigmoidActivationBackprop<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
															 dZ.data_device.get(),
															 Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform sigmoid back propagation");

	return dZ;
}
