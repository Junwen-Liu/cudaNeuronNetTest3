#include <map>
#include <sstream>
#include <string>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>

#include "neural_network.hh"
#include "layers/linear_layer.hh"
#include "layers/relu_activation.hh"
#include "layers/sigmoid_activation.hh"
#include "nn_utils/nn_exception.hh"
#include "nn_utils/bce_cost.hh"

#include "coordinates_dataset.hh"
#include <iostream>

#include "cuda_dll.h"
#include "mainTest3.h"

using namespace std;
using std::map;
using std::pair;
using std::vector;
using std::stringstream;

float computeAccuracy(const Matrix& predictions, const Matrix& targets);

void JavaHashMapToJNIHashMap(JNIEnv *env, jobject hashMap, std::map<std::string, std::string>& mapOut) {
	// Get the Map's entry Set.
	jclass mapClass = env->FindClass("java/util/Map");
	if (mapClass == NULL) {
	  return;
	}
	jmethodID entrySet =
	  env->GetMethodID(mapClass, "entrySet", "()Ljava/util/Set;");
	if (entrySet == NULL) {
	  return;
	}
	jobject set = env->CallObjectMethod(hashMap, entrySet);
	if (set == NULL) {
	  return;
	}
	// Obtain an iterator over the Set
	jclass setClass = env->FindClass("java/util/Set");
	if (setClass == NULL) {
	  return;
	}
	jmethodID iterator =
	  env->GetMethodID(setClass, "iterator", "()Ljava/util/Iterator;");
	if (iterator == NULL) {
	  return;
	}
	jobject iter = env->CallObjectMethod(set, iterator);
	if (iter == NULL) {
	  return;
	}
	// Get the Iterator method IDs
	jclass iteratorClass = env->FindClass("java/util/Iterator");
	if (iteratorClass == NULL) {
	  return;
	}
	jmethodID hasNext = env->GetMethodID(iteratorClass, "hasNext", "()Z");
	if (hasNext == NULL) {
	  return;
	}
	jmethodID next =
	  env->GetMethodID(iteratorClass, "next", "()Ljava/lang/Object;");
	if (next == NULL) {
	  return;
	}
	// Get the Entry class method IDs
	jclass entryClass = env->FindClass("java/util/Map$Entry");
	if (entryClass == NULL) {
	  return;
	}
	jmethodID getKey =
	  env->GetMethodID(entryClass, "getKey", "()Ljava/lang/Object;");
	if (getKey == NULL) {
	  return;
	}
	jmethodID getValue =
	  env->GetMethodID(entryClass, "getValue", "()Ljava/lang/Object;");
	if (getValue == NULL) {
	  return;
	}
	// Iterate over the entry Set
	while (env->CallBooleanMethod(iter, hasNext)) {
	  jobject entry = env->CallObjectMethod(iter, next);
	  jstring key = (jstring) env->CallObjectMethod(entry, getKey);
	  jstring value = (jstring) env->CallObjectMethod(entry, getValue);
	  const char* keyStr = env->GetStringUTFChars(key, NULL);
	  if (!keyStr) {  // Out of memory
		return;
	  }
	  const char* valueStr = env->GetStringUTFChars(value, NULL);
	  if (!valueStr) {  // Out of memory
		env->ReleaseStringUTFChars(key, keyStr);
		return;
	  }
  
	  mapOut.insert(std::make_pair(string(keyStr), string(valueStr)));
  
	  env->DeleteLocalRef(entry);
	  env->ReleaseStringUTFChars(key, keyStr);
	  env->DeleteLocalRef(key);
	  env->ReleaseStringUTFChars(value, valueStr);
	  env->DeleteLocalRef(value);
	}

}


JNIEXPORT jboolean JNICALL Java_mainTest3_simpleNN(JNIEnv *env, jobject, jobject hashMap){
// JNIEXPORT jboolean JNICALL Java_mainTest_simpleNN(JNIEnv *, jobject, jint numOfLayers, jobjectArray layerSpecs);
//int mainTest(map<int, int*> inputMap) {

	//Todo: parse jobject to hashMap 
	map<string, string>stringMap;
	map<int, vector<string>>inputMap;
	JavaHashMapToJNIHashMap(env, hashMap, stringMap);

	//parse string hashMap to int Hashmap
	map<string,string>::iterator it;
	for(it = stringMap.begin(); it != stringMap.end(); it++)
	{
		//parse second string to vector
		vector<string> vect;
		std::stringstream ss(it->second);

		while(ss.good())
		{
			string substr;
			getline(ss, substr, ',');
			vect.push_back( substr);
		}
		inputMap.insert(std::pair<int, vector<string>>(stoi(it->first), vect));
	}

	srand( time(NULL) );

	//batch_size=100, number of batches=21, use 20 batches for training and 1 batch for testing(get accuracy score)
	CoordinatesDataset dataset(100, 21);
	BCECost bce_cost;

	NeuralNetwork nn;

	//map<int, vector<string>>::iterator it;

	for (int i = 0; i < inputMap.size(); i++)
	{
		// it = inputMap.find(i);
		// if(it!=inputMap.end())
		vector<string> val = inputMap[i];

		switch(stoi(val[0])){
			case 0: 
				nn.addLayer(new LinearLayer("linear_"+i, Shape(stoi(val[1]), stoi(val[2]))));
				cout << "A linearLayer_" << i << " layer has been added in Nueron Net\n";
				break;
			case 1: 
				nn.addLayer(new ReLUActivation("relu_"+i));
				cout << "A relu_" << i << " layer has been added in Nueron Net\n";
				break;
			case 2:
				nn.addLayer(new SigmoidActivation("sigmoid_output_"+i));
				cout << "A sigmoid_" << i << " layer has been added in Nueron Net\n";
				break;
			default:
				break;
		}

	}

	cout << "The Nueron Net has completed construction, start the training for 1000 epochs...\n";

	////linear layer with 2 input neuron and 30 output/hidden neurons 
	//nn.addLayer(new LinearLayer("linear_1", Shape(2, 30)));
	//nn.addLayer(new ReLUActivation("relu_1"));
	////linear layer with 30 input neurons and 1 output neuron
	//nn.addLayer(new LinearLayer("linear_2", Shape(30, 1)));
	//nn.addLayer(new SigmoidActivation("sigmoid_output"));

	// network training
	Matrix Y;
	for (int epoch = 0; epoch < 1001; epoch++) {
		float cost = 0.0;

		for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) {
			//get training batches as input for forward
			Y = nn.forward(dataset.getBatches().at(batch));
			//get label/target batches as input for backprop
			nn.backprop(Y, dataset.getTargets().at(batch));
			cost += bce_cost.cost(Y, dataset.getTargets().at(batch));
		}

		if (epoch % 100 == 0) {
			std::cout 	<< "Epoch: " << epoch
						<< ", Cost: " << cost / dataset.getNumOfBatches()
						<< std::endl;
		}
	}

	// compute accuracy
	Y = nn.forward(dataset.getBatches().at(dataset.getNumOfBatches() - 1));
	Y.copyDeviceToHost();

	float accuracy = computeAccuracy(
			Y, dataset.getTargets().at(dataset.getNumOfBatches() - 1));
	std::cout 	<< "Accuracy: " << accuracy << std::endl;

	return true;
	}

//count number of correctly predicted values and divide it by the size of output vector
float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
	int m = predictions.shape.x;
	int correct_predictions = 0;

	for (int i = 0; i < m; i++) {
		float prediction = predictions[i] > 0.5 ? 1 : 0;
		if (prediction == targets[i]) {
			correct_predictions++;
		}
	}

	return static_cast<float>(correct_predictions) / m;
}

// Testing purpose in visual studio
// void main() {
// 	map<int, int*> testdata;
// 	int layer1[] = { 0,2,30 };
// 	testdata[0] = layer1;
// 	int layer2[] = { 1 };
// 	testdata[1] = layer2;
// 	int layer3[] = { 0,30,1 };
// 	testdata[2] = layer3;
// 	int layer4[] = { 2 };
// 	testdata[3] = layer4;

// 	int test = mainTest(testdata);
// 	cout << "final result is" << test;
// }

