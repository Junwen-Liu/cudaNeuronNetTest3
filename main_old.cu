
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <iostream>
//#include <time.h>
//
//#include "neural_network.hh"
//#include "layers/linear_layer.hh"
//#include "layers/relu_activation.hh"
//#include "layers/sigmoid_activation.hh"
//#include "nn_utils/nn_exception.hh"
//#include "nn_utils/bce_cost.hh"
//
//#include "coordinates_dataset.hh"
//
//float computeAccuracy(const Matrix& predictions, const Matrix& targets);
//
//int main() {
//
//	srand( time(NULL) );
//
//	//batch_size=100, number of batches=21, use 20 batches for training and 1 batch for testing(get accuracy score)
//	CoordinatesDataset dataset(100, 21);
//	BCECost bce_cost;
//
//	NeuralNetwork nn;
//	//linear layer with 2 input neuron and 30 output/hidden neurons 
//	nn.addLayer(new LinearLayer("linear_1", Shape(2, 30)));
//	nn.addLayer(new ReLUActivation("relu_1"));
//	//linear layer with 30 input neurons and 1 output neuron
//	nn.addLayer(new LinearLayer("linear_2", Shape(30, 1)));
//	nn.addLayer(new SigmoidActivation("sigmoid_output"));
//
//	// network training
//	Matrix Y;
//	for (int epoch = 0; epoch < 1001; epoch++) {
//		float cost = 0.0;
//
//		for (int batch = 0; batch < dataset.getNumOfBatches() - 1; batch++) {
//			//get training batches as input for forward
//			Y = nn.forward(dataset.getBatches().at(batch));
//			//get label/target batches as input for backprop
//			nn.backprop(Y, dataset.getTargets().at(batch));
//			cost += bce_cost.cost(Y, dataset.getTargets().at(batch));
//		}
//
//		if (epoch % 100 == 0) {
//			std::cout 	<< "Epoch: " << epoch
//						<< ", Cost: " << cost / dataset.getNumOfBatches()
//						<< std::endl;
//		}
//	}
//
//	// compute accuracy
//	Y = nn.forward(dataset.getBatches().at(dataset.getNumOfBatches() - 1));
//	Y.copyDeviceToHost();
//
//	float accuracy = computeAccuracy(
//			Y, dataset.getTargets().at(dataset.getNumOfBatches() - 1));
//	std::cout 	<< "Accuracy: " << accuracy << std::endl;
//
//	return 0;
//}
//
////count number of correctly predicted values and divide it by the size of output vector
//float computeAccuracy(const Matrix& predictions, const Matrix& targets) {
//	int m = predictions.shape.x;
//	int correct_predictions = 0;
//
//	for (int i = 0; i < m; i++) {
//		float prediction = predictions[i] > 0.5 ? 1 : 0;
//		if (prediction == targets[i]) {
//			correct_predictions++;
//		}
//	}
//
//	return static_cast<float>(correct_predictions) / m;
//}
